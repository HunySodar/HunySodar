
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define M 1000
#define N 500
#define K 1000
#define MAX 100000

#define block_size 16

//使用统一内存，不需要显式的内存拷贝
__managed__ int a[M*N], b[N*K], c[M*K];

//共享内存，是一块可以被同一block中的所有线程访问的内存
__global__ void matix_gpu(int *a, int *b, int *c, int m, int n, int k)
{
    //定义共享内存
    __shared__ int a_tile[block_size][block_size];
    __shared__ int b_tile[block_size][block_size];
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    int tmp = 0;
    for (int i = 0;i<=n/block_size;i++)
    {
        //将a的一块数据拷贝到共享内存中
        //对于结果矩阵c[y][x],需要将a[y][threadIdx.x]保存到共享内存a_tile[threadIdx.y][threadIdx.x]中
        //完成一个分块矩阵的计算之后，将a[y][threadIdx.x+block_size]保存到共享内存a_tile[threadIdx.y][threadIdx.x]中
        if (i*block_size+threadIdx.x<n && y<m)
        {
            a_tile[threadIdx.y][threadIdx.x] = a[y*n+i*block_size+threadIdx.x];
        }
        else
        {
            a_tile[threadIdx.y][threadIdx.x] = 0;
        }
        //将b的一块数据拷贝到共享内存中
        //对于结果矩阵c[y][x],需要将b[threadIdx.y][x]保存到共享内存b_tile[threadIdx.y][threadIdx.x]中
        if (i*block_size+threadIdx.y<n && x<k)
        {
            b_tile[threadIdx.y][threadIdx.x] = b[(i*block_size+threadIdx.y)*k+x];
        }
        else
        {
            b_tile[threadIdx.y][threadIdx.x] = 0;
        }
        //同步，等待所有线程将数据拷贝到共享内存中
        __syncthreads();
        //对于结果矩阵的每个元素，需要找到对应的行和列，然后进行相乘求和
        //c[i][j] = a[i][0]*b[0][j] + a[i][1]*b[1][j] + ... + a[i][n-1]*b[n-1][j]
        for (int j = 0; j < block_size; j++)
        {
            tmp += a_tile[threadIdx.y][j] * b_tile[j][threadIdx.x];
        }
        //同步，等待所有线程完成计算
        __syncthreads();
    }
    if (y<m && x<k)
    {
        c[y*k+x] = tmp;
    }
}

//矩阵转置函数，使用共享内存用作缓存
//因为全局内存存在合并访存，所以使用共享内存用作缓存，先从全局内存中读取数据a[3][2]到共享内存a[3][2]对应位置中，
//然后从共享内存中读取数据，在每个block中进行考虑
__global__ void trans(int in[M][N], int out[N][M])
{
    //定义共享内存
    __shared__ int tile[block_size][block_size+1];
    //计算线程的全局索引
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x<N && y<M)
    {
        //将数据从输入矩阵中读取到共享内存中
        tile[threadIdx.y][threadIdx.x] = in[y][x];
    }
    
    //同步，等待所有线程将数据拷贝到共享内存中
    __syncthreads();
    //计算转置矩阵的索引
    //进行转置之后，block看作一个整体，它的索引发生改变为blockIdx.x, blockIdx.y
    //在每个block中找到对应的位置，为blockIdx.x*blockDim.x+threadIdx.y, blockIdx.y*blockDim.y+threadIdx.x
    int x1 = blockIdx.x * blockDim.x + threadIdx.y;
    int y1 = blockIdx.y * blockDim.y + threadIdx.x;
    if(x1<M && y1<N)
    {
        //将数据从共享内存中读取到输出矩阵中
        out[y1][x1] = tile[threadIdx.x][threadIdx.y];
    }
}

//对于cpu计算矩阵乘法，需要遍历结果矩阵m*k的每一个元素
void matrix_cpu(int *a, int *b, int *c, int m, int n, int k)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j <k; j++)
        {
            int sum = 0;
            //对于结果矩阵的每个元素，需要找到对应的行和列，然后进行相乘求和
            //c[i][j] = a[i][0]*b[0][j] + a[i][1]*b[1][j] + ... + a[i][n-1]*b[n-1][j]
            for (int l = 0; l < n; l++)
            {
                sum += a[i*n+l] * b[l*k+j];
            }
            c[i*k+j] = sum;
        }
    }
}

//求一个数组的所有元素的和
__global__ void sum_gpu(int *a, int *sum, int n)
{
    //定义共享内存
    __shared__ int tile[block_size];
    //在这里假设线程的数量远小于数组的长度，在读入数据到共享内存的时候，每个线程读入多个元素
    //计算线程的全局索引
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int shared_tmp = 0;
    for(int idx=x;idx<n;idx+=blockDim.x*gridDim.x)
    {
        shared_tmp += a[idx];
    }
    tile[threadIdx.x] = shared_tmp;
    
    //同步，等待所有线程将数据拷贝到共享内存中
    __syncthreads();
    //对于每个block中的数据进行求和
    int tmp = 0;
    for (int i = block_size/2; i >= 1; i/=2)
    {
        if (threadIdx.x<i)
        {
            tmp += tile[threadIdx.x+i];
        }
        __syncthreads();
        if(threadIdx.x<i)
        {
            tile[threadIdx.x] = tmp;
        }
    }
    //将每个block中的结果保存到全局内存中,此时每个block中的tile[0]保存的是每个block中的和
    if (threadIdx.x==0)
    {
        //对于同一内存写入时，需要使用原子操作，因为多个线程可能同时写入
        atomicAdd(sum, tile[0]);
    }
}

int main()
{
    for (int i = 0; i <N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            a[i*M+j] = rand()%100;
        }
    }
    for (int i = 0; i <K; i++)
    {
        for (int j = 0; j < N; j++)
        {
            b[i*N+j] = rand()%100;
        }
    }

    //使用二维线程块
    dim3 block(block_size, block_size);
    dim3 grid((K+block.x-1)/block.x, (M+block.y-1)/block.y);
}
